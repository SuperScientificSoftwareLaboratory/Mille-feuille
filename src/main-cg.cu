#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include "./biio2.0/src/biio.h"
#include "Mille-feuille-cg.h"

extern "C" void cg_solve_reduce(int *RowPtr, int *ColIdx, MAT_VAL_TYPE *Val, MAT_VAL_LOW_TYPE *Val_Low, double *x, double *b, int n, int *iter, int maxiter, double threshold, char *filename, int nnzR, int ori)
{
    struct timeval t1, t2, t3, t4,t5,t6,t7,t8,t9,t10;
    int rowA = n;
    int colA = ori;
    rowA = (rowA / BLOCK_SIZE) * BLOCK_SIZE;
    Tile_matrix *matrix = (Tile_matrix *)malloc(sizeof(Tile_matrix));
    Tile_create(matrix,
                rowA, colA, nnzR,
                RowPtr,
                ColIdx,
                Val,
                Val_Low);
    int num_seg = ceil((double)rowA / BLOCK_SIZE);
    // num_seg += 1;
    //printf("rowA=%d colA=%d\n", rowA, colA);
    int tilenum = matrix->tilenum;
    int *ptroffset1 = (int *)malloc(sizeof(int) * tilenum);
    int *ptroffset2 = (int *)malloc(sizeof(int) * tilenum);
    memset(ptroffset1, 0, sizeof(int) * tilenum);
    memset(ptroffset2, 0, sizeof(int) * tilenum);
    MAT_VAL_TYPE *y_golden = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * rowA);
    MAT_VAL_TYPE *y = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * n);
    memset(x, 0, sizeof(double) * n);
    memset(y, 0, sizeof(MAT_VAL_TYPE) * n);
    int rowblkblock = 0;
    unsigned int *blkcoostylerowidx;
    int *blkcoostylerowidx_colstart;
    int *blkcoostylerowidx_colstop;
    int device_id = 0;
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);
    blockspmv_cpu(matrix,
                 ptroffset1,
                 ptroffset2,
                 &rowblkblock,
                 &blkcoostylerowidx,
                 &blkcoostylerowidx_colstart,
                 &blkcoostylerowidx_colstop,
                 rowA, colA, nnzR,
                 RowPtr,
                 ColIdx,
                 Val,
                 x,
                 y,
                 y_golden);
    int tilem = matrix->tilem;
    int tilen = matrix->tilen;
    MAT_PTR_TYPE *tile_ptr = matrix->tile_ptr;
    int *tile_columnidx = matrix->tile_columnidx;
    int *tile_nnz = matrix->tile_nnz;
    int *csr_offset = matrix->csr_offset;
    int *csrptr_offset = matrix->csrptr_offset;
    MAT_VAL_TYPE *Blockcsr_Val = matrix->Blockcsr_Val;
    MAT_VAL_LOW_TYPE *Blockcsr_Val_Low = matrix->Blockcsr_Val_Low;
    unsigned char *Tile_csr_Col = matrix->Tile_csr_Col;
    unsigned char *csr_compressedIdx = matrix->csr_compressedIdx;
    unsigned char *Blockcsr_Ptr = matrix->Blockcsr_Ptr;
    int csrsize = matrix->csrsize;
    int csrptrlen = matrix->csrptrlen;

    int csr_csize = csrsize % 2 == 0 ? csrsize / 2 : csrsize / 2 + 1;

    MAT_PTR_TYPE *d_tile_ptr;
    int *d_tile_columnidx;
    int *tile_rowidx = (int *)malloc(sizeof(int) * tilenum);
    memset(tile_rowidx, 0, sizeof(int) * tilenum);
    int *d_tile_rowidx;
    hipMalloc((void **)&d_tile_rowidx, tilenum * sizeof(int));
    hipMalloc((void **)&d_tile_ptr, (tilem + 1) * sizeof(MAT_PTR_TYPE));
    hipMalloc((void **)&d_tile_columnidx, tilenum * sizeof(int));

    hipMemcpy(d_tile_ptr, tile_ptr, (tilem + 1) * sizeof(MAT_PTR_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_tile_columnidx, tile_columnidx, tilenum * sizeof(int), hipMemcpyHostToDevice);

    // CSR
    unsigned char *d_csr_compressedIdx = (unsigned char *)malloc((csr_csize) * sizeof(unsigned char));
    MAT_VAL_TYPE *d_Blockcsr_Val;
    unsigned char *d_Blockcsr_Ptr;

    hipMalloc((void **)&d_csr_compressedIdx, (csr_csize) * sizeof(unsigned char));
    hipMalloc((void **)&d_Blockcsr_Val, (csrsize) * sizeof(MAT_VAL_TYPE));
    hipMalloc((void **)&d_Blockcsr_Ptr, (csrptrlen) * sizeof(unsigned char));

    hipMemcpy(d_csr_compressedIdx, csr_compressedIdx, (csr_csize) * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockcsr_Val, Blockcsr_Val, (csrsize) * sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockcsr_Ptr, Blockcsr_Ptr, (csrptrlen) * sizeof(unsigned char), hipMemcpyHostToDevice);



    unsigned int *d_blkcoostylerowidx;
    int *d_blkcoostylerowidx_colstart;
    int *d_blkcoostylerowidx_colstop;

    hipMalloc((void **)&d_blkcoostylerowidx, rowblkblock * sizeof(unsigned int));
    hipMalloc((void **)&d_blkcoostylerowidx_colstart, rowblkblock * sizeof(int));
    hipMalloc((void **)&d_blkcoostylerowidx_colstop, rowblkblock * sizeof(int));

    hipMemcpy(d_blkcoostylerowidx, blkcoostylerowidx, rowblkblock * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_blkcoostylerowidx_colstart, blkcoostylerowidx_colstart, rowblkblock * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_blkcoostylerowidx_colstop, blkcoostylerowidx_colstop, rowblkblock * sizeof(int), hipMemcpyHostToDevice);

    int *d_ptroffset1;
    int *d_ptroffset2;

    hipMalloc((void **)&d_ptroffset1, tilenum * sizeof(int));
    hipMalloc((void **)&d_ptroffset2, tilenum * sizeof(int));
    hipMemcpy(d_ptroffset1, ptroffset1, tilenum * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ptroffset2, ptroffset2, tilenum * sizeof(int), hipMemcpyHostToDevice);

    // x and y
    MAT_VAL_TYPE *d_x;
    MAT_VAL_TYPE *d_y;

    hipMalloc((void **)&d_x, rowA * sizeof(MAT_VAL_TYPE));
    hipMalloc((void **)&d_y, rowA * sizeof(MAT_VAL_TYPE));
    int num_threads = WARP_PER_BLOCK * WARP_SIZE;
    int num_blocks = ceil((double)rowblkblock / (double)(num_threads / WARP_SIZE));

    double *k_b, *k_x, *k_r, *k_d, *k_q, *k_s;
    double *k_alpha, *k_snew, *k_beta, *k_sold, *k_s0;
    double t, s0, snew;
    double *k_val;
    int iterations = 0;

    hipMalloc((void **)&k_b, sizeof(double) * (n));
    hipMemcpy(k_b, b, sizeof(double) * (n), hipMemcpyHostToDevice);
    hipMalloc((void **)&k_val, sizeof(double) * (nnzR));
    hipMemcpy(k_val, Val, sizeof(double) * (nnzR), hipMemcpyHostToDevice);

    hipMalloc((void **)&k_x, sizeof(double) * (n));
    hipMalloc((void **)&k_r, sizeof(double) * (n + 1));
    hipMalloc((void **)&k_d, sizeof(double) * (n + 1));
    hipMalloc((void **)&k_q, sizeof(double) * (n));
    hipMalloc((void **)&k_s, sizeof(double) * (n));
    hipMalloc((void **)&k_alpha, sizeof(double));
    hipMalloc((void **)&k_snew, sizeof(double) * NUM_BLOCKS);
    hipMalloc((void **)&k_sold, sizeof(double));
    hipMalloc((void **)&k_beta, sizeof(double));
    hipMalloc((void **)&k_s0, sizeof(double));
    double *r = (double *)malloc(sizeof(double) * (n + 1));
    memset(r, 0, sizeof(double) * (n + 1));
    double alpha;

    dim3 BlockDim(256);
    dim3 GridDim((n/256+1));

    veczero<<<1, BlockDim>>>(n, k_x);
    // r=b-Ax (r=b since x=0), and d=M^(-1)r
    hipMemcpy(k_r, k_b, sizeof(double) * (n), hipMemcpyDeviceToDevice);
    hipMemset(k_s0, 0, sizeof(double));
    sdot2_2<<<GridDim, BlockDim>>>(k_r, k_r, k_s0, n);
    hipMemcpy(k_d, k_r, sizeof(double) * (n + 1), hipMemcpyDeviceToDevice);
    //  snew = s0
    scalarassign(k_snew, k_s0);
    // Copy snew and s0 back to host so that host can evaluate stopping condition
    hipMemcpy(&snew, k_snew, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&s0, k_s0, sizeof(double), hipMemcpyDeviceToHost);
    double time_spmv = 0;

    int csroffset = 0;
    int csrcount = 0;
    int *nonzero_row_new = (int *)malloc(sizeof(int) * (tilenum + 1));
    memset(nonzero_row_new, 0, sizeof(int) * (tilenum + 1));
    gettimeofday(&t5, NULL);
#pragma omp parallel for
    for (int blki = 0; blki < tilem; blki++)
    {
        int rowlength = blki == tilem - 1 ? rowA - (tilem - 1) * BLOCK_SIZE : BLOCK_SIZE;
        for (int blkj = matrix->tile_ptr[blki]; blkj < matrix->tile_ptr[blki + 1]; blkj++)
        {
            csrcount = ptroffset2[blkj];
            tile_rowidx[blkj] = blki;
            for (int ri = 0; ri < rowlength; ri++)
            {
                int stop = ri == rowlength - 1 ? (matrix->blknnz[blkj + 1] - matrix->blknnz[blkj]) : matrix->Blockcsr_Ptr[ri + 1 + csrcount];
                if (stop != matrix->Blockcsr_Ptr[csrcount + ri])
                {
                    nonzero_row_new[blkj] += 1;
                }
            }
            nonzero_row_new[blkj] += 1;
        }
    }
    exclusive_scan(nonzero_row_new, tilenum + 1);
    int cnt_non_new = nonzero_row_new[tilenum];
    unsigned char *blockrowid_new = (unsigned char *)malloc(sizeof(unsigned char) * (cnt_non_new + 1));
    memset(blockrowid_new, 0, sizeof(unsigned char) * (cnt_non_new + 1));
    unsigned char *blockcsr_ptr_new = (unsigned char *)malloc(sizeof(unsigned char) * (cnt_non_new + 1));
    memset(blockcsr_ptr_new, 0, sizeof(unsigned char) * (cnt_non_new + 1));
    int csrcount_new1 = 0;
#pragma omp parallel for
    for (int blki = 0; blki < tilem; blki++)
    {
        int rowlength = blki == tilem - 1 ? rowA - (tilem - 1) * BLOCK_SIZE : BLOCK_SIZE;
        for (int blkj = matrix->tile_ptr[blki]; blkj < matrix->tile_ptr[blki + 1]; blkj++)
        {
            csrcount = ptroffset2[blkj];
            csrcount_new1 = nonzero_row_new[blkj];
            int fl = 0;
            for (int ri = 0; ri < rowlength; ri++)
            {
                int stop = ri == rowlength - 1 ? (matrix->blknnz[blkj + 1] - matrix->blknnz[blkj]) : matrix->Blockcsr_Ptr[ri + 1 + csrcount];
                if (ri == 0)
                {
                    blockrowid_new[csrcount_new1 + fl] = ri;
                    blockcsr_ptr_new[csrcount_new1 + fl] = 0;
                    fl++;
                }
                if (stop != matrix->Blockcsr_Ptr[csrcount + ri])
                {
                    blockrowid_new[csrcount_new1 + fl] = ri;
                    blockcsr_ptr_new[csrcount_new1 + fl] = stop;
                    fl++;
                }
            }
        }
    }
    gettimeofday(&t6, NULL);
    double time_format= (t6.tv_sec - t5.tv_sec) * 1000.0 + (t6.tv_usec - t5.tv_usec) / 1000.0;
    double pro_cnt=0.0;
    double time_dot=0.0;
    double time_axpy=0.0;
    unsigned char *d_blockrowid_new;
    unsigned char *d_blockcsr_ptr_new;
    int *d_nonzero_row_new;
    unsigned char *d_Tile_csr_Col;
    hipMalloc((void **)&d_blockrowid_new, sizeof(unsigned char) * (cnt_non_new + 1));
    hipMalloc((void **)&d_blockcsr_ptr_new, sizeof(unsigned char) * (cnt_non_new + 1));
    hipMalloc((void **)&d_nonzero_row_new, sizeof(int) * (tilenum + 1));
    hipMalloc((void **)&d_Tile_csr_Col, sizeof(unsigned char) * (matrix->csrsize));
    hipMemcpy(d_blockrowid_new, blockrowid_new, sizeof(unsigned char) * (cnt_non_new + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_blockcsr_ptr_new, blockcsr_ptr_new, sizeof(unsigned char) * (cnt_non_new + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_nonzero_row_new, nonzero_row_new, sizeof(int) * (tilenum + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_Tile_csr_Col, Tile_csr_Col, sizeof(unsigned char) * (matrix->csrsize), hipMemcpyHostToDevice);
    hipMemcpy(d_tile_rowidx, tile_rowidx, sizeof(int) * (tilenum), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    gettimeofday(&t1, NULL);
    while (iterations < 10000)
    //while (iterations < 1000 && sqrt(snew) > epsilon)
    {
        hipMemset(k_q, 0, n * sizeof(double));

        stir_spmv_cuda_kernel_newcsr<<<num_blocks, num_threads>>>(tilem, tilen, rowA, colA, nnzR,
                                                                              d_tile_ptr, d_tile_columnidx,
                                                                              d_csr_compressedIdx, d_Blockcsr_Val, d_Blockcsr_Ptr,
                                                                              d_ptroffset1, d_ptroffset2,
                                                                              rowblkblock, d_blkcoostylerowidx, d_blkcoostylerowidx_colstart, d_blkcoostylerowidx_colstop,
                                                                              k_d, k_q, d_blockrowid_new, d_blockcsr_ptr_new, d_nonzero_row_new, d_Tile_csr_Col);
   
        hipMemset(k_alpha, 0, sizeof(double));
        sdot2_2<<<GridDim, BlockDim>>>(k_d, k_q, k_alpha, n);
  
        scalardiv<<<1, 1>>>(k_snew, k_alpha, k_alpha);


        axpy<<<GridDim, BlockDim>>>(n, k_alpha, k_d, k_x, k_x);
  
        ymax<<<GridDim, BlockDim>>>(n, k_alpha, k_q, k_r);
        scalarassign(k_sold, k_snew);
        
        hipMemset(k_snew, 0, sizeof(double));
        sdot2_2<<<GridDim, BlockDim>>>(k_r, k_r, k_snew, n);
       
        scalardiv<<<1, 1>>>(k_snew, k_sold, k_beta);

       
        axpy<<<GridDim, BlockDim>>>(n, k_beta, k_d, k_r, k_d);
        
        hipMemcpy(&snew, k_snew, sizeof(double), hipMemcpyDeviceToHost);
       
        iterations++;
    }
    hipDeviceSynchronize();
    hipMemcpy(x, k_x, sizeof(double) * (n), hipMemcpyDeviceToHost);
    gettimeofday(&t2, NULL);
    double time_cg = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    printf("time_cg=%lf ms\n",time_cg/100);
    double time_total = time_spmv + time_dot + time_axpy;
    double *b_new = (double *)malloc(sizeof(double) * n);
    memset(b_new, 0, sizeof(double) * n);
    for (int blki = 0; blki < tilem; blki++)
    {
        for (int ri = 0; ri < BLOCK_SIZE; ri++)
        {
            b_new[blki * BLOCK_SIZE + ri] = 0;
        }
        for (int blkj = matrix->tile_ptr[blki]; blkj < matrix->tile_ptr[blki + 1]; blkj++)
        {
            int csrcolidx = tile_columnidx[blkj];
            int x_offset = csrcolidx * BLOCK_SIZE;
            csroffset = matrix->csr_offset[blkj];
            for (int ri = nonzero_row_new[blkj]; ri < nonzero_row_new[blkj + 1]; ri++)
            {
                double sum_new = 0;
                int ro = blockrowid_new[ri + 1];
                for (int rj = blockcsr_ptr_new[ri]; rj < blockcsr_ptr_new[ri + 1]; rj++)
                {
                    int csrcol = Tile_csr_Col[csroffset + rj];
                    sum_new += x[x_offset + csrcol] * matrix->Blockcsr_Val[csroffset + rj];
                }
                b_new[blki * BLOCK_SIZE + ro] += sum_new;
            }
        }
    }
    double sum = 0;
    for (int i = 0; i < n; i++)
    {
        double r = b_new[i] - b[i];
        sum = sum + (r * r);
    }
    double sum_ori = 0;
    for (int i = 0; i < n; i++)
    {
        sum_ori = sum_ori + (b[i] * b[i]);
    }
    double l2_norm = sqrt(sum) / sqrt(sum_ori);
    char *s = (char *)malloc(sizeof(char) * 200);
    sprintf(s, "%d,%.3f,%d,%e,%e\n", 100, time_cg/100, nnzR, l2_norm,sqrt(snew));
    FILE *file1 = fopen("data/cg_performance.csv", "a");
    if (file1 == NULL)
    {
        printf("open error!\n");
        return;
    }
    fwrite(filename, strlen(filename), 1, file1);
    fwrite(",", strlen(","), 1, file1);
    fwrite(s, strlen(s), 1, file1);
    free(s);
    hipFree(k_val);
    hipFree(k_b);
    hipFree(k_x);
    hipFree(k_r);
    hipFree(k_d);
    hipFree(k_q);
    hipFree(k_alpha);
    hipFree(k_snew);
    hipFree(k_sold);
    hipFree(k_beta);
    hipFree(k_s0);
    hipFree(d_tile_ptr);
    hipFree(d_tile_columnidx);
    hipFree(d_csr_compressedIdx);
    hipFree(d_Blockcsr_Val);
    hipFree(d_Blockcsr_Ptr);
    hipFree(d_blkcoostylerowidx);
    hipFree(d_blkcoostylerowidx_colstart);
    hipFree(d_blkcoostylerowidx_colstop);
    hipFree(d_ptroffset1);
    hipFree(d_ptroffset2);
    hipFree(d_x);
    hipFree(d_y);
    free(matrix);
    free(ptroffset1);
    free(ptroffset2);
    free(y_golden);
    free(y);
    free(blkcoostylerowidx);
    free(blkcoostylerowidx_colstart);
    free(blkcoostylerowidx_colstop);
    free(tile_ptr);
    free(tile_columnidx);
    free(tile_nnz);
    free(csr_offset);
    free(csrptr_offset);
    free(Blockcsr_Val);
    free(Blockcsr_Val_Low);
    free(csr_compressedIdx);
    free(Blockcsr_Ptr);
}


extern "C" void cg_solve_sync(int *RowPtr, int *ColIdx, MAT_VAL_TYPE *Val, MAT_VAL_LOW_TYPE *Val_Low, double *x, double *b, int n, int *iter, int maxiter, double threshold, char *filename, int nnzR, int ori)
{
    struct timeval t1, t2, t3, t4, t5, t6;
    int rowA = n;
    int colA = ori;
    rowA = (rowA / BLOCK_SIZE) * BLOCK_SIZE;
    Tile_matrix *matrix = (Tile_matrix *)malloc(sizeof(Tile_matrix));
    Tile_create(matrix,
                rowA, colA, nnzR,
                RowPtr,
                ColIdx,
                Val,
                Val_Low);
    int num_seg = ceil((double)rowA / BLOCK_SIZE);
    int tilenum = matrix->tilenum;
    int *ptroffset1 = (int *)malloc(sizeof(int) * tilenum);
    int *ptroffset2 = (int *)malloc(sizeof(int) * tilenum);
    memset(ptroffset1, 0, sizeof(int) * tilenum);
    memset(ptroffset2, 0, sizeof(int) * tilenum);
    MAT_VAL_TYPE *y_golden = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * rowA);
    MAT_VAL_TYPE *y = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * n);
    memset(x, 0, sizeof(double) * n);
    memset(y, 0, sizeof(MAT_VAL_TYPE) * n);
    int rowblkblock = 0;
    unsigned int *blkcoostylerowidx;
    int *blkcoostylerowidx_colstart;
    int *blkcoostylerowidx_colstop;
    int device_id = 0;
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);
    blockspmv_cpu(matrix,
                  ptroffset1,
                  ptroffset2,
                  &rowblkblock,
                  &blkcoostylerowidx,
                  &blkcoostylerowidx_colstart,
                  &blkcoostylerowidx_colstop,
                  rowA, colA, nnzR,
                  RowPtr,
                  ColIdx,
                  Val,
                  x,
                  y,
                  y_golden);
    int tilem = matrix->tilem;
    int tilen = matrix->tilen;
    MAT_PTR_TYPE *tile_ptr = matrix->tile_ptr;
    int *tile_columnidx = matrix->tile_columnidx;
    int *tile_nnz = matrix->tile_nnz;
    int *csr_offset = matrix->csr_offset;
    int *csrptr_offset = matrix->csrptr_offset;
    MAT_VAL_TYPE *Blockcsr_Val = matrix->Blockcsr_Val;
    MAT_VAL_LOW_TYPE *Blockcsr_Val_Low = matrix->Blockcsr_Val_Low;
    unsigned char *Tile_csr_Col = matrix->Tile_csr_Col;
    unsigned char *csr_compressedIdx = matrix->csr_compressedIdx;
    unsigned char *Blockcsr_Ptr = matrix->Blockcsr_Ptr;
    int csrsize = matrix->csrsize;
    int csrptrlen = matrix->csrptrlen;

    int csr_csize = csrsize % 2 == 0 ? csrsize / 2 : csrsize / 2 + 1;

    MAT_PTR_TYPE *d_tile_ptr;
    int *d_tile_columnidx;
    int *tile_rowidx = (int *)malloc(sizeof(int) * tilenum);
    memset(tile_rowidx, 0, sizeof(int) * tilenum);
    int *d_tile_rowidx;
    hipMalloc((void **)&d_tile_rowidx, tilenum * sizeof(int));
    hipMalloc((void **)&d_tile_ptr, (tilem + 1) * sizeof(MAT_PTR_TYPE));
    hipMalloc((void **)&d_tile_columnidx, tilenum * sizeof(int));

    hipMemcpy(d_tile_ptr, tile_ptr, (tilem + 1) * sizeof(MAT_PTR_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_tile_columnidx, tile_columnidx, tilenum * sizeof(int), hipMemcpyHostToDevice);
    int *tile_columnidx_new=(int *)malloc(sizeof(int)*tilenum);
    memset(tile_columnidx_new,0,sizeof(int)*tilenum);
    // CSR
    unsigned char *d_csr_compressedIdx = (unsigned char *)malloc((csr_csize) * sizeof(unsigned char));
    MAT_VAL_TYPE *d_Blockcsr_Val;
    unsigned char *d_Blockcsr_Ptr;

    hipMalloc((void **)&d_csr_compressedIdx, (csr_csize) * sizeof(unsigned char));
    hipMalloc((void **)&d_Blockcsr_Val, (csrsize) * sizeof(MAT_VAL_TYPE));
    hipMalloc((void **)&d_Blockcsr_Ptr, (csrptrlen) * sizeof(unsigned char));

    hipMemcpy(d_csr_compressedIdx, csr_compressedIdx, (csr_csize) * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockcsr_Val, Blockcsr_Val, (csrsize) * sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockcsr_Ptr, Blockcsr_Ptr, (csrptrlen) * sizeof(unsigned char), hipMemcpyHostToDevice);

    unsigned int *d_blkcoostylerowidx;
    int *d_blkcoostylerowidx_colstart;
    int *d_blkcoostylerowidx_colstop;

    hipMalloc((void **)&d_blkcoostylerowidx, rowblkblock * sizeof(unsigned int));
    hipMalloc((void **)&d_blkcoostylerowidx_colstart, rowblkblock * sizeof(int));
    hipMalloc((void **)&d_blkcoostylerowidx_colstop, rowblkblock * sizeof(int));

    hipMemcpy(d_blkcoostylerowidx, blkcoostylerowidx, rowblkblock * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_blkcoostylerowidx_colstart, blkcoostylerowidx_colstart, rowblkblock * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_blkcoostylerowidx_colstop, blkcoostylerowidx_colstop, rowblkblock * sizeof(int), hipMemcpyHostToDevice);

    int *d_ptroffset1;
    int *d_ptroffset2;

    hipMalloc((void **)&d_ptroffset1, tilenum * sizeof(int));
    hipMalloc((void **)&d_ptroffset2, tilenum * sizeof(int));
    hipMemcpy(d_ptroffset1, ptroffset1, tilenum * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ptroffset2, ptroffset2, tilenum * sizeof(int), hipMemcpyHostToDevice);

    // x and y
    MAT_VAL_TYPE *d_x;
    MAT_VAL_TYPE *d_y;

    hipMalloc((void **)&d_x, rowA * sizeof(MAT_VAL_TYPE));
    hipMalloc((void **)&d_y, rowA * sizeof(MAT_VAL_TYPE));
    int num_threads = WARP_PER_BLOCK * WARP_SIZE;
    int num_blocks = ceil((double)rowblkblock / (double)(num_threads / WARP_SIZE));
    int num_blocks_new = ceil((double)(tilem) / (double)(num_threads / WARP_SIZE));
    double *k_b, *k_x, *k_r, *k_d, *k_q, *k_s;
    double *k_alpha, *k_snew, *k_beta, *k_sold, *k_s0;
    double t, s0, snew;
    double alpha;
    double *k_val;
    int iterations = 0;

    hipMalloc((void **)&k_b, sizeof(double) * (n));
    hipMemcpy(k_b, b, sizeof(double) * (n), hipMemcpyHostToDevice);
    hipMalloc((void **)&k_val, sizeof(double) * (nnzR));
    hipMemcpy(k_val, Val, sizeof(double) * (nnzR), hipMemcpyHostToDevice);

    hipMalloc((void **)&k_x, sizeof(double) * (n));
    hipMalloc((void **)&k_r, sizeof(double) * (n + 1));
    hipMalloc((void **)&k_d, sizeof(double) * (n + 1));
    hipMalloc((void **)&k_q, sizeof(double) * (n));
    hipMalloc((void **)&k_s, sizeof(double) * (n));
    hipMalloc((void **)&k_alpha, sizeof(double));
    hipMalloc((void **)&k_snew, sizeof(double));
    hipMalloc((void **)&k_sold, sizeof(double));
    hipMalloc((void **)&k_beta, sizeof(double));
    hipMalloc((void **)&k_s0, sizeof(double));
    double *r = (double *)malloc(sizeof(double) * (n + 1));
    memset(r, 0, sizeof(double) * (n + 1));

    dim3 BlockDim(256);
    dim3 GridDim((n / 256 + 1));

    veczero<<<1, BlockDim>>>(n, k_x);
    // r=b-Ax (r=b since x=0), and d=M^(-1)r
    hipMemcpy(k_r, k_b, sizeof(double) * (n), hipMemcpyDeviceToDevice);
    hipMemset(k_s0, 0, sizeof(double));
    sdot2_2<<<GridDim, BlockDim>>>(k_r, k_r, k_s0, n);
    hipMemcpy(k_d, k_r, sizeof(double) * (n + 1), hipMemcpyDeviceToDevice);
    //  snew = s0
    scalarassign(k_snew, k_s0);
    // Copy snew and s0 back to host so that host can evaluate stopping condition
    hipMemcpy(&snew, k_snew, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&s0, k_s0, sizeof(double), hipMemcpyDeviceToHost);
    double time_spmv = 0;
    int csroffset = 0;
    int csrcount = 0;
    int *nonzero_row_new = (int *)malloc(sizeof(int) * (tilenum + 1));
    memset(nonzero_row_new, 0, sizeof(int) * (tilenum + 1));
    for (int blki = 0; blki < tilem; blki++)
    {
        int rowlength = blki == tilem - 1 ? rowA - (tilem - 1) * BLOCK_SIZE : BLOCK_SIZE;
        for (int blkj = matrix->tile_ptr[blki]; blkj < matrix->tile_ptr[blki + 1]; blkj++)
        {
            csrcount = ptroffset2[blkj];
            tile_rowidx[blkj] = blki;
            for (int ri = 0; ri < rowlength; ri++)
            {
                int stop = ri == rowlength - 1 ? (matrix->blknnz[blkj + 1] - matrix->blknnz[blkj]) : matrix->Blockcsr_Ptr[ri + 1 + csrcount];
                if (stop != matrix->Blockcsr_Ptr[csrcount + ri])
                {
                    nonzero_row_new[blkj] += 1;
                }
            }
            nonzero_row_new[blkj] += 1;
        }
    }
    exclusive_scan(nonzero_row_new, tilenum + 1);
    int cnt_non_new = nonzero_row_new[tilenum];
    unsigned char *blockrowid_new = (unsigned char *)malloc(sizeof(unsigned char) * (cnt_non_new + 1));
    memset(blockrowid_new, 0, sizeof(unsigned char) * (cnt_non_new + 1));
    unsigned char *blockcsr_ptr_new = (unsigned char *)malloc(sizeof(unsigned char) * (cnt_non_new + 1));
    memset(blockcsr_ptr_new, 0, sizeof(unsigned char) * (cnt_non_new + 1));
    int csrcount_new1 = 0;
    int *block_signal = (int *)malloc(sizeof(int) * (tilem + 1));
    memset(block_signal, 0, sizeof(int) * (tilem + 1)); 
    for (int blki = 0; blki < tilem; blki++)
    {
        int rowlength = blki == tilem - 1 ? rowA - (tilem - 1) * BLOCK_SIZE : BLOCK_SIZE;
        block_signal[blki] = matrix->tile_ptr[blki + 1] - matrix->tile_ptr[blki];
        for (int blkj = matrix->tile_ptr[blki]; blkj < matrix->tile_ptr[blki + 1]; blkj++)
        {
            csrcount = ptroffset2[blkj];
            csrcount_new1 = nonzero_row_new[blkj];
            int fl = 0;
            for (int ri = 0; ri < rowlength; ri++)
            {
                int stop = ri == rowlength - 1 ? (matrix->blknnz[blkj + 1] - matrix->blknnz[blkj]) : matrix->Blockcsr_Ptr[ri + 1 + csrcount];
                if (ri == 0)
                {
                    blockrowid_new[csrcount_new1 + fl] = ri;
                    blockcsr_ptr_new[csrcount_new1 + fl] = 0;
                    fl++;
                }
                if (stop != matrix->Blockcsr_Ptr[csrcount + ri])
                {
                    blockrowid_new[csrcount_new1 + fl] = ri;
                    blockcsr_ptr_new[csrcount_new1 + fl] = stop;
                    fl++;
                }
            }
        }
    }

    
    int *non_each_block = (int *)malloc(sizeof(int) * (tilenum + 1));        
    int *non_each_block_offset = (int *)malloc(sizeof(int) * (tilenum + 1));
    int *row_each_block = (int *)malloc(sizeof(int) * (tilenum + 1));       
    int *index_each_block = (int *)malloc(sizeof(int) * (tilenum + 1));      
    memset(non_each_block, 0, sizeof(int) * (tilenum + 1));
    memset(non_each_block_offset, 0, sizeof(int) * (tilenum + 1));
    memset(row_each_block, 0, sizeof(int) * (tilenum + 1));
    memset(index_each_block, 0, sizeof(int) * (tilenum + 1));
    int nnz_total = 0;
    for (int blki = 0; blki < tilem; blki++)
    {
        for (int blkj = tile_ptr[blki]; blkj < tile_ptr[blki + 1]; blkj++)
        {
            non_each_block[blkj] = matrix->blknnz[blkj + 1] - matrix->blknnz[blkj];
            nnz_total += non_each_block[blkj];
            row_each_block[blkj] = blki;
            index_each_block[blkj] = blkj;
        }
    }
    int *row_each_block_new = (int *)malloc(sizeof(int) * (tilenum + 1));  
    int *index_each_block_new = (int *)malloc(sizeof(int) * (tilenum + 1)); 
    int *non_each_block_new = (int *)malloc(sizeof(int) * (tilenum + 1));
    memset(row_each_block_new, 0, sizeof(int) * (tilenum + 1));
    memset(index_each_block_new, 0, sizeof(int) * (tilenum + 1));
    memset(non_each_block_new, 0, sizeof(int) * (tilenum + 1));
   
    int each_block_nnz = 16;
    
    int cnt = 0;
    int balance_row = 0;
    int index = 1;
    
    int block_per_warp=180;
   
    int i = 0;
    int j = tilenum - 1;
    int step = 0;
    int cnt_block1=0;
    int nnz_list[12]={16,32,64,96,128,256,512,1024,2048,4096,nnzR/6912};
    while(1)
    {
    for(int k=0;k<12;k++)
    {
    each_block_nnz=nnz_list[k];
    i = 0;
    j = tilenum - 1;
    cnt = 0;
    index = 1;
    step = 0;
    cnt_block1=0;
    while (i < j)
    {
        if (((non_each_block[i] + cnt) < each_block_nnz)&&((cnt_block1+1)<block_per_warp))
        {
            cnt += non_each_block[i];
            i++;
            cnt_block1++;
        }
        else if (((non_each_block[i] + cnt) >= each_block_nnz)||((cnt_block1+1)>=block_per_warp))
        {
            i++;
            index++;
            cnt = 0;
            cnt_block1=0;
        }
        if (((non_each_block[j] + cnt) < each_block_nnz)&&((cnt_block1+1)<block_per_warp))
        {
            cnt += non_each_block[j];
            j--;
            cnt_block1++;
        }
        else if (((non_each_block[j] + cnt) >= each_block_nnz)||((cnt_block1+1)>=block_per_warp))
        {
            j--;
            index++;
            cnt = 0;
            cnt_block1=0;
        }
    }
    if(index<6912)
    break;
    }
    if(index<6912)
    break;
    block_per_warp=block_per_warp*2;
    }    
    int vector_each_warp_16;
    int vector_total_16;
    int vector_each_warp_32;
    int vector_total_32;
    if (index < tilem)
    {
        vector_each_warp_16 = ceil((double)(tilem) / (double)(index));
        vector_total_16 = tilem / vector_each_warp_16;
        int tilem_32 = ceil((double)tilem / 2);
        vector_each_warp_32 = vector_each_warp_16*2;
        vector_total_32 = tilem_32 / vector_each_warp_32;
        vector_total_32 = (vector_total_32/WARP_PER_BLOCK+1)*WARP_PER_BLOCK;
    }
    if (index > 6912||index==0||tilem==0)
        return;
    int *balance_tile_ptr_new = (int *)malloc(sizeof(int) * (index + 1));
    memset(balance_tile_ptr_new, 0, sizeof(int) * (index + 1));
    int *balance_tile_ptr_shared_end = (int *)malloc(sizeof(int) * (index + 1));
    memset(balance_tile_ptr_shared_end, 0, sizeof(int) * (index + 1));
    i = 0;
    j = tilenum - 1;
    cnt = 0;
    index = 1;
    step = 0;
    cnt_block1=0;
    while (i < j)
    {
        if (((non_each_block[i] + cnt) < each_block_nnz)&&((cnt_block1+1)<block_per_warp))
        {
            cnt += non_each_block[i];
            index_each_block_new[step] = index_each_block[i];
            row_each_block_new[step] = row_each_block[i];
            non_each_block_new[step] = non_each_block[i];
            i++;
            step++;
            cnt_block1++;
        }
        else if (((non_each_block[i] + cnt) >= each_block_nnz)||((cnt_block1+1)>=block_per_warp))
        {
            index_each_block_new[step] = index_each_block[i];
            row_each_block_new[step] = row_each_block[i];
            non_each_block_new[step] = non_each_block[i];
            i++;
            step++;
            balance_tile_ptr_new[index] = step;
            index++;
            cnt = 0;
            cnt_block1=0;
        }
         if (((non_each_block[j] + cnt) < each_block_nnz)&&((cnt_block1+1)<block_per_warp))
        {
            cnt += non_each_block[j];
            index_each_block_new[step] = index_each_block[j];
            row_each_block_new[step] = row_each_block[j];
            non_each_block_new[step] = non_each_block[j];
            j--;
            step++;
            cnt_block1++;
        }
        else if (((non_each_block[j] + cnt) >= each_block_nnz)||((cnt_block1+1)>=block_per_warp))
        {
            index_each_block_new[step] = index_each_block[j];
            row_each_block_new[step] = row_each_block[j];
            non_each_block_new[step] = non_each_block[j];
            j--;
            step++;
            balance_tile_ptr_new[index] = step;
            index++;
            cnt = 0;
            cnt_block1=0;
        }
        if (i == j)
        {
            index_each_block_new[step] = index_each_block[j];
            row_each_block_new[step] = row_each_block[j];
            non_each_block_new[step] = non_each_block[j];
            step++;
            balance_tile_ptr_new[index] = step;
        }
        if (i > j)
        {
            index_each_block_new[step] = index_each_block[j];
            row_each_block_new[step] = row_each_block[j];
            non_each_block_new[step] = non_each_block[j];
            balance_tile_ptr_new[index] = step;
        }
    }
    
    int *d_balance_tile_ptr_new;
    hipMalloc((void **)&d_balance_tile_ptr_new, sizeof(int) * (index + 1));
    hipMemcpy(d_balance_tile_ptr_new, balance_tile_ptr_new, sizeof(int) * (index + 1), hipMemcpyHostToDevice);
    int *d_row_each_block;
    int *d_index_each_block;
    hipMalloc((void **)&d_row_each_block, sizeof(int) * (tilenum + 1));
    hipMalloc((void **)&d_index_each_block, sizeof(int) * (tilenum + 1));
    hipMemcpy(d_row_each_block, row_each_block_new, sizeof(int) * (tilenum + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_index_each_block, index_each_block_new, sizeof(int) * (tilenum + 1), hipMemcpyHostToDevice);
     

    // int cnt_block = 0;
    // int cnt_nnz = 0;

    // for (int i = 0; i <= index; i++)
    // {
    //     balance_tile_ptr_shared_end[i] = balance_tile_ptr_new[i];
    // }
    // int cnt_nnz_shared=0;
    // int shared_nnz_each_block=256;
    // for (int i = 0; i < index; i++)
    // {
    //     cnt_nnz = 0;
    //     cnt_nnz_shared=0;
    //     for (int j = balance_tile_ptr_new[i]; j < balance_tile_ptr_new[i + 1]; j++)
    //     {
    //         int blkj=index_each_block_new[j];
    //         if (j == balance_tile_ptr_new[i])
    //             non_each_block_offset[j] = 0;
    //         cnt_nnz += non_each_block_new[j];
    //         cnt_block++;
    //         if (j != balance_tile_ptr_new[i] && cnt_nnz <=shared_nnz_each_block)
    //         {
    //             cnt_nnz_shared+=non_each_block_new[j - 1];
    //             non_each_block_offset[j] = non_each_block_new[j - 1];
    //             non_each_block_offset[j] += non_each_block_offset[j - 1];
    //         }
    //         if (cnt_nnz > shared_nnz_each_block)
    //         {
    //             balance_tile_ptr_shared_end[i + 1] = j;
    //             break;
    //         }
    //     }
    // }
    // cnt_nnz_shared = 0;
    // int cnt_nnz_total = 0;
    // int shared_num=0;
    // for (int i = 0; i < index; i++)
    // {
    //     cnt_nnz = 0;
    //     cnt_nnz_shared = 0;
    //     cnt_nnz_total = 0;
    //     for (int j = balance_tile_ptr_new[i]; j < balance_tile_ptr_new[i + 1]; j++)
    //     {
    //         cnt_nnz_total += non_each_block_new[j];
    //     }
    //     for (int j = balance_tile_ptr_new[i]; j < balance_tile_ptr_shared_end[i + 1]; j++)
    //     {
    //         cnt_nnz_shared += non_each_block_new[j];
    //         shared_num++;
    //     }
    //     for (int j = balance_tile_ptr_shared_end[i + 1]; j < balance_tile_ptr_new[i + 1]; j++)
    //     {
    //         cnt_nnz += non_each_block_new[j];
    //     }
    // }

    // int *d_non_each_block_offset;
    // hipMalloc((void **)&d_non_each_block_offset, sizeof(int) * (tilenum + 1));
    // hipMemcpy(d_non_each_block_offset, non_each_block_offset, sizeof(int) * (tilenum + 1), hipMemcpyHostToDevice);

    // int *d_balance_tile_ptr_shared_end;
    // hipMalloc((void **)&d_balance_tile_ptr_shared_end, sizeof(int) * (index + 1));
    // hipMemcpy(d_balance_tile_ptr_shared_end, balance_tile_ptr_shared_end, sizeof(int) * (index + 1), hipMemcpyHostToDevice);
    
    int *d_block_signal;
    hipMalloc((void **)&d_block_signal, sizeof(int) * (tilem + 1));
    int *signal_dot;
    hipMalloc((void **)&signal_dot, sizeof(int));
    int *signal_final;
    hipMalloc((void **)&signal_final, sizeof(int));
    int *signal_final1;
    hipMalloc((void **)&signal_final1, sizeof(int));
    hipMemset(signal_final1, 0, sizeof(int));
    double *k_threshold;
    hipMalloc((void **)&k_threshold, sizeof(double));
    int *d_ori_block_signal;
    hipMalloc((void **)&d_ori_block_signal, sizeof(int) * (tilem + 1));
    hipMemcpy(d_block_signal, block_signal, sizeof(int) * (tilem + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_ori_block_signal, block_signal, sizeof(int) * (tilem + 1), hipMemcpyHostToDevice);
    double pro_cnt = 0.0;
    unsigned char *d_blockrowid_new;
    unsigned char *d_blockcsr_ptr_new;
    int *d_nonzero_row_new;
    unsigned char *d_Tile_csr_Col;
    hipMalloc((void **)&d_blockrowid_new, sizeof(unsigned char) * (cnt_non_new + 1));
    hipMalloc((void **)&d_blockcsr_ptr_new, sizeof(unsigned char) * (cnt_non_new + 1));
    hipMalloc((void **)&d_nonzero_row_new, sizeof(int) * (tilenum + 1));
    hipMalloc((void **)&d_Tile_csr_Col, sizeof(unsigned char) * (matrix->csrsize));
    hipMemcpy(d_blockrowid_new, blockrowid_new, sizeof(unsigned char) * (cnt_non_new + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_blockcsr_ptr_new, blockcsr_ptr_new, sizeof(unsigned char) * (cnt_non_new + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_nonzero_row_new, nonzero_row_new, sizeof(int) * (tilenum + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_Tile_csr_Col, Tile_csr_Col, sizeof(unsigned char) * (matrix->csrsize), hipMemcpyHostToDevice);
    hipMemcpy(d_tile_rowidx, tile_rowidx, sizeof(int) * (tilenum), hipMemcpyHostToDevice);
    threshold = epsilon * epsilon * s0;
    double *k_x_new;
    int *d_block_signal_new;
    int *d_ori_block_signal_new;
    double *k_q_new;
    double *k_d_new;
    double *k_r_new;
    hipMemcpy(k_threshold, &threshold, sizeof(double), hipMemcpyHostToDevice);
    gettimeofday(&t1, NULL);
    {
        if (index < tilem)
        {
            int num_blocks_nnz_balance = ceil((double)(index) / (double)(num_threads / WARP_SIZE));
            hipMemset(d_block_signal,0,sizeof(int) * (tilem + 1));
            if(vector_each_warp_32*vector_total_32*32>rowA)
            {
                rowA=vector_each_warp_32*vector_total_32*32;
            }
            int tilem_new=rowA/BLOCK_SIZE;
            hipMalloc((void **)&d_block_signal_new, sizeof(int) * (tilem_new + 1));
            hipMemset(d_block_signal_new,0,sizeof(int) * (tilem_new + 1));
            hipMalloc((void **)&d_ori_block_signal_new, sizeof(int) * (tilem_new + 1));
            hipMemset(d_ori_block_signal_new,0,sizeof(int) * (tilem_new + 1));
            hipMemcpy(d_ori_block_signal_new, block_signal, sizeof(int) * (tilem + 1), hipMemcpyHostToDevice);
            hipMalloc((void **)&k_q_new, sizeof(double) * (rowA));
            hipMalloc((void **)&k_d_new, sizeof(double) * (rowA));
            hipMemset(k_d_new, 0, (rowA) * sizeof(double));
            hipMemcpy(k_d_new, k_r, sizeof(double) * (n), hipMemcpyDeviceToDevice);
            hipMalloc((void **)&k_r_new, sizeof(double) * (rowA));
            hipMemset(k_r_new, 0, (rowA) * sizeof(double));
            hipMemcpy(k_r_new, k_r, sizeof(double) * (n), hipMemcpyDeviceToDevice);
            hipMalloc((void **)&k_x_new, sizeof(double) * (rowA));
            hipMemset(k_x_new, 0, (rowA) * sizeof(double));
            hipMemcpy(k_x_new, k_x, sizeof(double) * (n), hipMemcpyDeviceToDevice);
            hipDeviceSynchronize();
            gettimeofday(&t3, NULL);
            stir_spmv_cuda_kernel_newcsr_nnz_balance_below_tilem_32_block_reduce<<<num_blocks_nnz_balance, num_threads>>>(tilem, tilenum, rowA, colA, nnzR,
                                                                                                             d_tile_ptr, d_tile_columnidx,
                                                                                                             d_csr_compressedIdx, d_Blockcsr_Val, d_Blockcsr_Ptr,
                                                                                                             d_ptroffset1, d_ptroffset2,
                                                                                                             rowblkblock, d_blkcoostylerowidx, d_blkcoostylerowidx_colstart, d_blkcoostylerowidx_colstop,
                                                                                                             k_d_new, k_q_new, d_blockrowid_new, d_blockcsr_ptr_new, d_nonzero_row_new, d_Tile_csr_Col, d_block_signal_new,
                                                                                                             signal_dot, signal_final, signal_final1, d_ori_block_signal_new,
                                                                                                             k_alpha, k_snew, k_x_new, k_r_new, k_sold, k_beta, k_threshold,
                                                                                                             d_balance_tile_ptr_new, d_row_each_block, d_index_each_block, index,
                                                                                                             vector_each_warp_32, vector_total_32);
            hipDeviceSynchronize();
            gettimeofday(&t4, NULL);
            time_spmv += (t4.tv_sec - t3.tv_sec) * 1000.0 + (t4.tv_usec - t3.tv_usec) / 1000.0;
        }
        else
        {
            if(index==tilem)
            index=tilem+1;
            hipMemset(d_block_signal,0,sizeof(int) * (tilem + 1));
            int num_blocks_nnz_balance = ceil((double)(index) / (double)(num_threads / WARP_SIZE));
            int tilem_new=(tilem/WARP_PER_BLOCK+2)*WARP_PER_BLOCK;
            int re_size=(tilem_new)*BLOCK_SIZE;
            hipMalloc((void **)&d_block_signal_new, sizeof(int) * (tilem_new + 1));
            hipMemset(d_block_signal_new,0,sizeof(int) * (tilem_new + 1));
            hipMalloc((void **)&d_ori_block_signal_new, sizeof(int) * (tilem_new + 1));
            hipMemset(d_ori_block_signal_new,0,sizeof(int) * (tilem_new + 1));
            hipMemcpy(d_ori_block_signal_new, block_signal, sizeof(int) * (tilem + 1), hipMemcpyHostToDevice);
            hipMalloc((void **)&k_q_new, sizeof(double) * re_size);
            hipMalloc((void **)&k_d_new, sizeof(double) * re_size);
            hipMemset(k_d_new, 0,  re_size* sizeof(double));
            hipMemcpy(k_d_new, k_r, sizeof(double) * (n), hipMemcpyDeviceToDevice);
            hipMalloc((void **)&k_r_new, sizeof(double) * re_size);
            hipMemset(k_r_new, 0, re_size * sizeof(double));
            hipMemcpy(k_r_new, k_r, sizeof(double) * (n), hipMemcpyDeviceToDevice);
            hipMalloc((void **)&k_x_new, sizeof(double) * re_size);
            hipMemset(k_x_new, 0, re_size * sizeof(double));
            hipMemcpy(k_x_new, k_x, sizeof(double) * (n), hipMemcpyDeviceToDevice);
            hipDeviceSynchronize();
            gettimeofday(&t3, NULL);
            stir_spmv_cuda_kernel_newcsr_nnz_balance_redce_block<<<num_blocks_nnz_balance, num_threads>>>(tilem_new, tilenum, rowA, colA, nnzR,
                                                                                              d_tile_ptr, d_tile_columnidx,
                                                                                              d_csr_compressedIdx, d_Blockcsr_Val, d_Blockcsr_Ptr,
                                                                                              d_ptroffset1, d_ptroffset2,
                                                                                              rowblkblock, d_blkcoostylerowidx, d_blkcoostylerowidx_colstart, d_blkcoostylerowidx_colstop,
                                                                                              k_d_new, k_q_new, d_blockrowid_new, d_blockcsr_ptr_new, d_nonzero_row_new, d_Tile_csr_Col, d_block_signal_new,
                                                                                              signal_dot, signal_final, signal_final1, d_ori_block_signal_new,
                                                                                              k_alpha, k_snew, k_x_new, k_r_new, k_sold, k_beta, k_threshold,
                                                                                              d_balance_tile_ptr_new, d_row_each_block, d_index_each_block, index);
            hipDeviceSynchronize();
            gettimeofday(&t4, NULL);
            time_spmv += (t4.tv_sec - t3.tv_sec) * 1000.0 + (t4.tv_usec - t3.tv_usec) / 1000.0;
            // stir_spmv_cuda_kernel_newcsr_nnz_balance_redce_block_shared_queue<<<num_blocks_nnz_balance, num_threads>>>(tilem_new, tilenum, rowA, colA, nnzR,
            //                                                                                   d_tile_ptr, d_tile_columnidx,
            //                                                                                   d_csr_compressedIdx, d_Blockcsr_Val, d_Blockcsr_Ptr,
            //                                                                                   d_ptroffset1, d_ptroffset2,
            //                                                                                   rowblkblock, d_blkcoostylerowidx, d_blkcoostylerowidx_colstart, d_blkcoostylerowidx_colstop,
            //                                                                                   k_d_new, k_q_new, d_blockrowid_new, d_blockcsr_ptr_new, d_nonzero_row_new, d_Tile_csr_Col, d_block_signal_new,
            //                                                                                   signal_dot, signal_final, signal_final1, d_ori_block_signal_new,
            //                                                                                   k_alpha, k_snew, k_x_new, k_r_new, k_sold, k_beta, k_threshold,
            //                                                                                   d_balance_tile_ptr_new, d_row_each_block, d_index_each_block, index, d_non_each_block_offset,d_balance_tile_ptr_shared_end,shared_num);
        }
        hipMemcpy(&snew, k_snew, sizeof(double), hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    hipMemcpy(x, k_x_new, sizeof(double) * (n), hipMemcpyDeviceToHost);
    double time_cg = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    printf("time_cg=%lf ms\n", time_spmv);
    double *b_new = (double *)malloc(sizeof(double) * n);
    memset(b_new, 0, sizeof(double) * n);
    for (int blki = 0; blki < tilem; blki++)
    {
        for (int blkj = matrix->tile_ptr[blki]; blkj < matrix->tile_ptr[blki + 1]; blkj++)
        {
            int csrcolidx = tile_columnidx[blkj];
            int x_offset = csrcolidx * BLOCK_SIZE;
            csroffset = matrix->csr_offset[blkj];
            for (int ri = nonzero_row_new[blkj]; ri < nonzero_row_new[blkj + 1]; ri++)
            {
                double sum_new = 0;
                int ro = blockrowid_new[ri + 1];
                for (int rj = blockcsr_ptr_new[ri]; rj < blockcsr_ptr_new[ri + 1]; rj++)
                {
                    int csrcol = Tile_csr_Col[csroffset + rj];
                    sum_new += x[x_offset + csrcol] * matrix->Blockcsr_Val[csroffset + rj];
                }
                b_new[blki * BLOCK_SIZE + ro] += sum_new;
            }
        }
    }
    double sum = 0;
    for (int i = 0; i < n; i++)
    {
        double r = b_new[i] - b[i];
        sum = sum + (r * r);
    }
    double sum_ori = 0;
    for (int i = 0; i < n; i++)
    {
        sum_ori = sum_ori + (b[i] * b[i]);
    }
    double l2_norm = sqrt(sum) / sqrt(sum_ori);
    char *s = (char *)malloc(sizeof(char) * 200);
    sprintf(s, "%d,%.3f,%d,%e,%e\n", 100, time_cg, nnzR, l2_norm,sqrt(snew));
    FILE *file1 = fopen("data/cg_performance.csv", "a");
    if (file1 == NULL)
    {
        printf("open error!\n");
        return;
    }
    fwrite(filename, strlen(filename), 1, file1);
    fwrite(",", strlen(","), 1, file1);
    fwrite(s, strlen(s), 1, file1);
    free(s);
    hipFree(k_val);
    hipFree(k_b);
    hipFree(k_x);
    hipFree(k_r);
    hipFree(k_d);
    hipFree(k_q);
    hipFree(k_alpha);
    hipFree(k_snew);
    hipFree(k_sold);
    hipFree(k_beta);
    hipFree(k_s0);
    hipFree(d_tile_ptr);
    hipFree(d_tile_columnidx);
    hipFree(d_csr_compressedIdx);
    hipFree(d_Blockcsr_Val);
    hipFree(d_Blockcsr_Ptr);
    hipFree(d_blkcoostylerowidx);
    hipFree(d_blkcoostylerowidx_colstart);
    hipFree(d_blkcoostylerowidx_colstop);
    hipFree(d_ptroffset1);
    hipFree(d_ptroffset2);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(k_x_new);
    hipFree(d_block_signal_new);
    hipFree(d_ori_block_signal_new);
    hipFree(k_q_new);
    hipFree(k_r_new);
    hipFree(k_d_new);
    free(matrix);
    free(ptroffset1);
    free(ptroffset2);
    free(y_golden);
    free(y);
    free(blkcoostylerowidx);
    free(blkcoostylerowidx_colstart);
    free(blkcoostylerowidx_colstop);
    free(tile_ptr);
    free(tile_columnidx);
    free(tile_nnz);
    free(csr_offset);
    free(csrptr_offset);
    free(Blockcsr_Val);
    free(Blockcsr_Val_Low);
    free(csr_compressedIdx);
    free(Blockcsr_Ptr);
}
extern "C" void cg_solve_inc(int *RowPtr, int *ColIdx, MAT_VAL_TYPE *Val, MAT_VAL_LOW_TYPE *Val_Low, double *x, double *b, int n, int *iter, int maxiter, double threshold, char *filename, int nnzR, int ori)
{
    struct timeval t1, t2, t3, t4, t5, t6;
    int rowA = n;
    int colA = ori;
    rowA = (rowA / BLOCK_SIZE) * BLOCK_SIZE;
    Tile_matrix *matrix = (Tile_matrix *)malloc(sizeof(Tile_matrix));
    Tile_create(matrix,
                rowA, colA, nnzR,
                RowPtr,
                ColIdx,
                Val,
                Val_Low);
    int num_seg = ceil((double)rowA / BLOCK_SIZE);
    int tilenum = matrix->tilenum;
    int *ptroffset1 = (int *)malloc(sizeof(int) * tilenum);
    int *ptroffset2 = (int *)malloc(sizeof(int) * tilenum);
    memset(ptroffset1, 0, sizeof(int) * tilenum);
    memset(ptroffset2, 0, sizeof(int) * tilenum);
    MAT_VAL_TYPE *y_golden = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * rowA);
    MAT_VAL_TYPE *y = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * n);
    memset(x, 0, sizeof(double) * n);
    memset(y, 0, sizeof(MAT_VAL_TYPE) * n);
    int rowblkblock = 0;
    unsigned int *blkcoostylerowidx;
    int *blkcoostylerowidx_colstart;
    int *blkcoostylerowidx_colstop;
    int device_id = 0;
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);
    blockspmv_cpu(matrix,
                  ptroffset1,
                  ptroffset2,
                  &rowblkblock,
                  &blkcoostylerowidx,
                  &blkcoostylerowidx_colstart,
                  &blkcoostylerowidx_colstop,
                  rowA, colA, nnzR,
                  RowPtr,
                  ColIdx,
                  Val,
                  x,
                  y,
                  y_golden);
    int tilem = matrix->tilem;
    int tilen = matrix->tilen;
    MAT_PTR_TYPE *tile_ptr = matrix->tile_ptr;
    int *tile_columnidx = matrix->tile_columnidx;
    int *tile_nnz = matrix->tile_nnz;
    int *csr_offset = matrix->csr_offset;
    int *csrptr_offset = matrix->csrptr_offset;
    MAT_VAL_TYPE *Blockcsr_Val = matrix->Blockcsr_Val;
    MAT_VAL_LOW_TYPE *Blockcsr_Val_Low = matrix->Blockcsr_Val_Low;
    unsigned char *Tile_csr_Col = matrix->Tile_csr_Col;
    unsigned char *csr_compressedIdx = matrix->csr_compressedIdx;
    unsigned char *Blockcsr_Ptr = matrix->Blockcsr_Ptr;
    int csrsize = matrix->csrsize;
    int csrptrlen = matrix->csrptrlen;

    int csr_csize = csrsize % 2 == 0 ? csrsize / 2 : csrsize / 2 + 1;

    MAT_PTR_TYPE *d_tile_ptr;
    int *d_tile_columnidx;
    int *tile_rowidx = (int *)malloc(sizeof(int) * tilenum);
    memset(tile_rowidx, 0, sizeof(int) * tilenum);
    int *d_tile_rowidx;
    hipMalloc((void **)&d_tile_rowidx, tilenum * sizeof(int));
    hipMalloc((void **)&d_tile_ptr, (tilem + 1) * sizeof(MAT_PTR_TYPE));
    hipMalloc((void **)&d_tile_columnidx, tilenum * sizeof(int));

    hipMemcpy(d_tile_ptr, tile_ptr, (tilem + 1) * sizeof(MAT_PTR_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_tile_columnidx, tile_columnidx, tilenum * sizeof(int), hipMemcpyHostToDevice);

    // CSR
    unsigned char *d_csr_compressedIdx = (unsigned char *)malloc((csr_csize) * sizeof(unsigned char));
    MAT_VAL_TYPE *d_Blockcsr_Val;
    unsigned char *d_Blockcsr_Ptr;

    hipMalloc((void **)&d_csr_compressedIdx, (csr_csize) * sizeof(unsigned char));
    hipMalloc((void **)&d_Blockcsr_Val, (csrsize) * sizeof(MAT_VAL_TYPE));
    hipMalloc((void **)&d_Blockcsr_Ptr, (csrptrlen) * sizeof(unsigned char));

    hipMemcpy(d_csr_compressedIdx, csr_compressedIdx, (csr_csize) * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockcsr_Val, Blockcsr_Val, (csrsize) * sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockcsr_Ptr, Blockcsr_Ptr, (csrptrlen) * sizeof(unsigned char), hipMemcpyHostToDevice);

    unsigned int *d_blkcoostylerowidx;
    int *d_blkcoostylerowidx_colstart;
    int *d_blkcoostylerowidx_colstop;

    hipMalloc((void **)&d_blkcoostylerowidx, rowblkblock * sizeof(unsigned int));
    hipMalloc((void **)&d_blkcoostylerowidx_colstart, rowblkblock * sizeof(int));
    hipMalloc((void **)&d_blkcoostylerowidx_colstop, rowblkblock * sizeof(int));

    hipMemcpy(d_blkcoostylerowidx, blkcoostylerowidx, rowblkblock * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_blkcoostylerowidx_colstart, blkcoostylerowidx_colstart, rowblkblock * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_blkcoostylerowidx_colstop, blkcoostylerowidx_colstop, rowblkblock * sizeof(int), hipMemcpyHostToDevice);

    int *d_ptroffset1;
    int *d_ptroffset2;

    hipMalloc((void **)&d_ptroffset1, tilenum * sizeof(int));
    hipMalloc((void **)&d_ptroffset2, tilenum * sizeof(int));
    hipMemcpy(d_ptroffset1, ptroffset1, tilenum * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ptroffset2, ptroffset2, tilenum * sizeof(int), hipMemcpyHostToDevice);

    // x and y
    MAT_VAL_TYPE *d_x;
    MAT_VAL_TYPE *d_y;

    hipMalloc((void **)&d_x, rowA * sizeof(MAT_VAL_TYPE));
    hipMalloc((void **)&d_y, rowA * sizeof(MAT_VAL_TYPE));
    int num_threads = WARP_PER_BLOCK * WARP_SIZE;
    int num_blocks = ceil((double)rowblkblock / (double)(num_threads / WARP_SIZE));
    int num_blocks_new = ceil((double)(tilem) / (double)(num_threads / WARP_SIZE));
    double *k_b, *k_x, *k_r, *k_d, *k_q, *k_s;
    double *k_alpha, *k_snew, *k_beta, *k_sold, *k_s0;
    double t, s0, snew;
    double alpha;
    double *k_val;
    int iterations = 0;

    hipMalloc((void **)&k_b, sizeof(double) * (n));
    hipMemcpy(k_b, b, sizeof(double) * (n), hipMemcpyHostToDevice);
    hipMalloc((void **)&k_val, sizeof(double) * (nnzR));
    hipMemcpy(k_val, Val, sizeof(double) * (nnzR), hipMemcpyHostToDevice);

    hipMalloc((void **)&k_x, sizeof(double) * (n));
    hipMalloc((void **)&k_r, sizeof(double) * (n + 1));
    hipMalloc((void **)&k_d, sizeof(double) * (n + 1));
    hipMalloc((void **)&k_q, sizeof(double) * (n));
    hipMalloc((void **)&k_s, sizeof(double) * (n));
    hipMalloc((void **)&k_alpha, sizeof(double));
    hipMalloc((void **)&k_snew, sizeof(double));
    hipMalloc((void **)&k_sold, sizeof(double));
    hipMalloc((void **)&k_beta, sizeof(double));
    hipMalloc((void **)&k_s0, sizeof(double));
    double *r = (double *)malloc(sizeof(double) * (n + 1));
    memset(r, 0, sizeof(double) * (n + 1));

    dim3 BlockDim(256);
    dim3 GridDim((n / 256 + 1));

    veczero<<<1, BlockDim>>>(n, k_x);
    // r=b-Ax (r=b since x=0), and d=M^(-1)r
    hipMemcpy(k_r, k_b, sizeof(double) * (n), hipMemcpyDeviceToDevice);
    hipMemset(k_s0, 0, sizeof(double));
    sdot2_2<<<GridDim, BlockDim>>>(k_r, k_r, k_s0, n);
    hipMemcpy(k_d, k_r, sizeof(double) * (n + 1), hipMemcpyDeviceToDevice);
    //  snew = s0
    scalarassign(k_snew, k_s0);
    // Copy snew and s0 back to host so that host can evaluate stopping condition
    hipMemcpy(&snew, k_snew, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&s0, k_s0, sizeof(double), hipMemcpyDeviceToHost);
    double time_spmv = 0;

    // tile_newcsr
    int csroffset = 0;
    int csrcount = 0;
    int *nonzero_row_new = (int *)malloc(sizeof(int) * (tilenum + 1));
    memset(nonzero_row_new, 0, sizeof(int) * (tilenum + 1));
#pragma omp parallel for
    for (int blki = 0; blki < tilem; blki++)
    {
        int rowlength = blki == tilem - 1 ? rowA - (tilem - 1) * BLOCK_SIZE : BLOCK_SIZE;
        for (int blkj = matrix->tile_ptr[blki]; blkj < matrix->tile_ptr[blki + 1]; blkj++)
        {
            csrcount = ptroffset2[blkj];
            tile_rowidx[blkj] = blki;
            for (int ri = 0; ri < rowlength; ri++)
            {
                int stop = ri == rowlength - 1 ? (matrix->blknnz[blkj + 1] - matrix->blknnz[blkj]) : matrix->Blockcsr_Ptr[ri + 1 + csrcount];
                if (stop != matrix->Blockcsr_Ptr[csrcount + ri])
                {
                    nonzero_row_new[blkj] += 1;
                }
            }
            nonzero_row_new[blkj] += 1;
        }
    }
    exclusive_scan(nonzero_row_new, tilenum + 1);
    int cnt_non_new = nonzero_row_new[tilenum];
    unsigned char *blockrowid_new = (unsigned char *)malloc(sizeof(unsigned char) * (cnt_non_new + 1));
    memset(blockrowid_new, 0, sizeof(unsigned char) * (cnt_non_new + 1));
    unsigned char *blockcsr_ptr_new = (unsigned char *)malloc(sizeof(unsigned char) * (cnt_non_new + 1));
    memset(blockcsr_ptr_new, 0, sizeof(unsigned char) * (cnt_non_new + 1));
    int csrcount_new1 = 0;
    int *block_signal = (int *)malloc(sizeof(int) * (tilem + 1));
    memset(block_signal, 0, sizeof(int) * (tilem + 1)); 
#pragma omp parallel for
    for (int blki = 0; blki < tilem; blki++)
    {
        int rowlength = blki == tilem - 1 ? rowA - (tilem - 1) * BLOCK_SIZE : BLOCK_SIZE;
        block_signal[blki] = matrix->tile_ptr[blki + 1] - matrix->tile_ptr[blki];
        for (int blkj = matrix->tile_ptr[blki]; blkj < matrix->tile_ptr[blki + 1]; blkj++)
        {
            csrcount = ptroffset2[blkj];
            csrcount_new1 = nonzero_row_new[blkj];
            int fl = 0;
            for (int ri = 0; ri < rowlength; ri++)
            {
                int stop = ri == rowlength - 1 ? (matrix->blknnz[blkj + 1] - matrix->blknnz[blkj]) : matrix->Blockcsr_Ptr[ri + 1 + csrcount];
                if (ri == 0)
                {
                    blockrowid_new[csrcount_new1 + fl] = ri;
                    blockcsr_ptr_new[csrcount_new1 + fl] = 0;
                    fl++;
                }
                if (stop != matrix->Blockcsr_Ptr[csrcount + ri])
                {
                    blockrowid_new[csrcount_new1 + fl] = ri;
                    blockcsr_ptr_new[csrcount_new1 + fl] = stop;
                    fl++;
                }
            }
        }
    }

    int *non_each_block = (int *)malloc(sizeof(int) * (tilenum + 1));        
    int *non_each_block_offset = (int *)malloc(sizeof(int) * (tilenum + 1));
    int *row_each_block = (int *)malloc(sizeof(int) * (tilenum + 1));        
    int *index_each_block = (int *)malloc(sizeof(int) * (tilenum + 1));     
    memset(non_each_block, 0, sizeof(int) * (tilenum + 1));
    memset(non_each_block_offset, 0, sizeof(int) * (tilenum + 1));
    memset(row_each_block, 0, sizeof(int) * (tilenum + 1));
    memset(index_each_block, 0, sizeof(int) * (tilenum + 1));
    int nnz_total = 0;
    for (int blki = 0; blki < tilem; blki++)
    {
        for (int blkj = tile_ptr[blki]; blkj < tile_ptr[blki + 1]; blkj++)
        {
            non_each_block[blkj] = matrix->blknnz[blkj + 1] - matrix->blknnz[blkj];
            nnz_total += non_each_block[blkj];
            row_each_block[blkj] = blki;
            index_each_block[blkj] = blkj;
        }
    }
   
    int *row_each_block_new = (int *)malloc(sizeof(int) * (tilenum + 1));   
    int *index_each_block_new = (int *)malloc(sizeof(int) * (tilenum + 1)); 
    int *non_each_block_new = (int *)malloc(sizeof(int) * (tilenum + 1));
    memset(row_each_block_new, 0, sizeof(int) * (tilenum + 1));
    memset(index_each_block_new, 0, sizeof(int) * (tilenum + 1));
    memset(non_each_block_new, 0, sizeof(int) * (tilenum + 1));

    int each_block_nnz = 16;
    int cnt = 0;
    int balance_row = 0;
    int index = 1;
    
    int i = 0;
    int j = tilenum - 1;
    cnt = 0;
    index = 1;
    int step = 0;
    int block_per_warp=180;
    int cnt_block1=0;
    int nnz_list[12]={16,32,64,96,128,256,512,1024,2048,4096,nnzR/6912};
    while(1)
    {
    for(int k=0;k<12;k++)
    {
    each_block_nnz=nnz_list[k];
    i = 0;
    j = tilenum - 1;
    cnt = 0;
    index = 1;
    step = 0;
    cnt_block1=0;
    while (i < j)
    {
        if ((non_each_block[i] + cnt) < each_block_nnz)
        {
            cnt += non_each_block[i];
            i++;
        }
        else if ((non_each_block[i] + cnt) >= each_block_nnz)
        {
            i++;
            index++;
            cnt = 0;
        }
        if ((non_each_block[j] + cnt) < each_block_nnz)
        {
            cnt += non_each_block[j];
            j--;
        }
        else if ((non_each_block[j] + cnt) >= each_block_nnz)
        {
            j--;
            index++;
            cnt = 0;
        }
    }
    if(index<6912)
    break;
    }
    if(index<6912)
    break;
    block_per_warp=block_per_warp*2;
    }
    int vector_each_warp_16;
    int vector_total_16;
    int vector_each_warp_32;
    int vector_total_32;
    if (index < tilem)
    {
        vector_each_warp_16 = ceil((double)(tilem) / (double)(index));
        vector_total_16 = tilem / vector_each_warp_16;
        int tilem_32 = ceil((double)tilem / 2);
        vector_each_warp_32 = vector_each_warp_16*2;
        vector_total_32 = tilem_32 / vector_each_warp_32;
        vector_total_32 = (vector_total_32/WARP_PER_BLOCK)*WARP_PER_BLOCK;
    }
    if (index > 6912)
        return;
    int *balance_tile_ptr_new = (int *)malloc(sizeof(int) * (index + 1));
    memset(balance_tile_ptr_new, 0, sizeof(int) * (index + 1));
    int *balance_tile_ptr_shared_end = (int *)malloc(sizeof(int) * (index + 1));
    memset(balance_tile_ptr_shared_end, 0, sizeof(int) * (index + 1));
    i = 0;
    j = tilenum - 1;
    cnt = 0;
    index = 1;
    step = 0;
    while (i < j)
    {
        if ((non_each_block[i] + cnt) < each_block_nnz)
        {
            cnt += non_each_block[i];
            index_each_block_new[step] = index_each_block[i];
            row_each_block_new[step] = row_each_block[i];
            non_each_block_new[step] = non_each_block[i];
            i++;
            step++;
        }
        else if ((non_each_block[i] + cnt) >= each_block_nnz)
        {
            index_each_block_new[step] = index_each_block[i];
            row_each_block_new[step] = row_each_block[i];
            non_each_block_new[step] = non_each_block[i];
            i++;
            step++;
            balance_tile_ptr_new[index] = step;
            index++;
            cnt = 0;
        }
        if ((non_each_block[j] + cnt) < each_block_nnz)
        {
            cnt += non_each_block[j];
            index_each_block_new[step] = index_each_block[j];
            row_each_block_new[step] = row_each_block[j];
            non_each_block_new[step] = non_each_block[j];
            j--;
            step++;
        }
        else if ((non_each_block[j] + cnt) >= each_block_nnz)
        {
            index_each_block_new[step] = index_each_block[j];
            row_each_block_new[step] = row_each_block[j];
            non_each_block_new[step] = non_each_block[j];
            j--;
            step++;
            balance_tile_ptr_new[index] = step;
            index++;
            cnt = 0;
        }
        if (i == j)
        {
            index_each_block_new[step] = index_each_block[j];
            row_each_block_new[step] = row_each_block[j];
            non_each_block_new[step] = non_each_block[j];
            step++;
            balance_tile_ptr_new[index] = step;
        }
        if (i > j)
        {
            index_each_block_new[step] = index_each_block[j];
            row_each_block_new[step] = row_each_block[j];
            non_each_block_new[step] = non_each_block[j];
            balance_tile_ptr_new[index] = step;
        }
    }
    int *d_balance_tile_ptr_new;
    hipMalloc((void **)&d_balance_tile_ptr_new, sizeof(int) * (index + 1));
    hipMemcpy(d_balance_tile_ptr_new, balance_tile_ptr_new, sizeof(int) * (index + 1), hipMemcpyHostToDevice);
    int *d_row_each_block;
    int *d_index_each_block;
    hipMalloc((void **)&d_row_each_block, sizeof(int) * (tilenum + 1));
    hipMalloc((void **)&d_index_each_block, sizeof(int) * (tilenum + 1));
    hipMemcpy(d_row_each_block, row_each_block_new, sizeof(int) * (tilenum + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_index_each_block, index_each_block_new, sizeof(int) * (tilenum + 1), hipMemcpyHostToDevice);
   


    int *d_block_signal;
    hipMalloc((void **)&d_block_signal, sizeof(int) * (tilem + 1));
    int *signal_dot;
    hipMalloc((void **)&signal_dot, sizeof(int));
    int *signal_final;
    hipMalloc((void **)&signal_final, sizeof(int));
    int *signal_final1;
    hipMalloc((void **)&signal_final1, sizeof(int));
    double *k_threshold;
    hipMalloc((void **)&k_threshold, sizeof(double));
    int *d_ori_block_signal;
    hipMalloc((void **)&d_ori_block_signal, sizeof(int) * (tilem + 1));
    hipMemcpy(d_block_signal, block_signal, sizeof(int) * (tilem + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_ori_block_signal, block_signal, sizeof(int) * (tilem + 1), hipMemcpyHostToDevice);
    double pro_cnt = 0.0;
    unsigned char *d_blockrowid_new;
    unsigned char *d_blockcsr_ptr_new;
    int *d_nonzero_row_new;
    unsigned char *d_Tile_csr_Col;
    hipMalloc((void **)&d_blockrowid_new, sizeof(unsigned char) * (cnt_non_new + 1));
    hipMalloc((void **)&d_blockcsr_ptr_new, sizeof(unsigned char) * (cnt_non_new + 1));
    hipMalloc((void **)&d_nonzero_row_new, sizeof(int) * (tilenum + 1));
    hipMalloc((void **)&d_Tile_csr_Col, sizeof(unsigned char) * (matrix->csrsize));
    hipMemcpy(d_blockrowid_new, blockrowid_new, sizeof(unsigned char) * (cnt_non_new + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_blockcsr_ptr_new, blockcsr_ptr_new, sizeof(unsigned char) * (cnt_non_new + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_nonzero_row_new, nonzero_row_new, sizeof(int) * (tilenum + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_Tile_csr_Col, Tile_csr_Col, sizeof(unsigned char) * (matrix->csrsize), hipMemcpyHostToDevice);
    hipMemcpy(d_tile_rowidx, tile_rowidx, sizeof(int) * (tilenum), hipMemcpyHostToDevice);
    threshold = epsilon * epsilon * s0;
    hipMemcpy(k_threshold, &threshold, sizeof(double), hipMemcpyHostToDevice);
    gettimeofday(&t1, NULL);
    {
        
        hipDeviceSynchronize();
        gettimeofday(&t3, NULL);
        if (index < tilem)
        {

            int num_blocks_nnz_balance = ceil((double)(index) / (double)(num_threads / WARP_SIZE));
            hipMemset(d_block_signal,0,sizeof(int) * (tilem + 1));
            stir_spmv_cuda_kernel_newcsr_nnz_balance_below_tilem_32_block_reduce<<<num_blocks_nnz_balance, num_threads>>>(tilem, tilenum, rowA, colA, nnzR,
                                                                                                             d_tile_ptr, d_tile_columnidx,
                                                                                                             d_csr_compressedIdx, d_Blockcsr_Val, d_Blockcsr_Ptr,
                                                                                                             d_ptroffset1, d_ptroffset2,
                                                                                                             rowblkblock, d_blkcoostylerowidx, d_blkcoostylerowidx_colstart, d_blkcoostylerowidx_colstop,
                                                                                                             k_d, k_q, d_blockrowid_new, d_blockcsr_ptr_new, d_nonzero_row_new, d_Tile_csr_Col, d_block_signal,
                                                                                                             signal_dot, signal_final, signal_final1, d_ori_block_signal,
                                                                                                             k_alpha, k_snew, k_x, k_r, k_sold, k_beta, k_threshold,
                                                                                                             d_balance_tile_ptr_new, d_row_each_block, d_index_each_block, index,
                                                                                                             vector_each_warp_32, vector_total_32);

        
        }
        else
        {
            int num_blocks_nnz_balance = ceil((double)(index) / (double)(num_threads / WARP_SIZE));
            stir_spmv_cuda_kernel_newcsr_nnz_balance<<<num_blocks_nnz_balance, num_threads>>>(tilem, tilenum, rowA, colA, nnzR,
                                                                                              d_tile_ptr, d_tile_columnidx,
                                                                                              d_csr_compressedIdx, d_Blockcsr_Val, d_Blockcsr_Ptr,
                                                                                              d_ptroffset1, d_ptroffset2,
                                                                                              rowblkblock, d_blkcoostylerowidx, d_blkcoostylerowidx_colstart, d_blkcoostylerowidx_colstop,
                                                                                              k_d, k_q, d_blockrowid_new, d_blockcsr_ptr_new, d_nonzero_row_new, d_Tile_csr_Col, d_block_signal,
                                                                                              signal_dot, signal_final, signal_final1, d_ori_block_signal,
                                                                                              k_alpha, k_snew, k_x, k_r, k_sold, k_beta, k_threshold,
                                                                                              d_balance_tile_ptr_new, d_row_each_block, d_index_each_block, index);
        }
        hipDeviceSynchronize();
        gettimeofday(&t4, NULL);
        time_spmv += (t4.tv_sec - t3.tv_sec) * 1000.0 + (t4.tv_usec - t3.tv_usec) / 1000.0;
        hipMemcpy(&snew, k_snew, sizeof(double), hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    hipMemcpy(x, k_x, sizeof(double) * (n), hipMemcpyDeviceToHost);
    double time_cg = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    printf("time_cg=%lf ms\n",time_spmv);
    double *b_new = (double *)malloc(sizeof(double) * n);
    memset(b_new, 0, sizeof(double) * n);
    for (int blki = 0; blki < tilem; blki++)
    {
        for (int blkj = matrix->tile_ptr[blki]; blkj < matrix->tile_ptr[blki + 1]; blkj++)
        {
            int csrcolidx = tile_columnidx[blkj];
            int x_offset = csrcolidx * BLOCK_SIZE;
            csroffset = matrix->csr_offset[blkj];
            for (int ri = nonzero_row_new[blkj]; ri < nonzero_row_new[blkj + 1]; ri++)
            {
                double sum_new = 0;
                int ro = blockrowid_new[ri + 1];
                for (int rj = blockcsr_ptr_new[ri]; rj < blockcsr_ptr_new[ri + 1]; rj++)
                {
                    int csrcol = Tile_csr_Col[csroffset + rj];
                    sum_new += x[x_offset + csrcol] * matrix->Blockcsr_Val[csroffset + rj];
                }
                b_new[blki * BLOCK_SIZE + ro] += sum_new;
            }
        }
    }
    double sum = 0;
    for (int i = 0; i < n; i++)
    {
        double r = b_new[i] - b[i];
        sum = sum + (r * r);
    }
    double sum_ori = 0;
    for (int i = 0; i < n; i++)
    {
        sum_ori = sum_ori + (b[i] * b[i]);
    }
    double l2_norm = sqrt(sum) / sqrt(sum_ori);
    char *s = (char *)malloc(sizeof(char) * 200);
    sprintf(s, "%d,%.3f,%d,%e,%e\n", 100, time_cg, nnzR, l2_norm,sqrt(snew));
    FILE *file1 = fopen("data/cg_performance.csv", "a");
    if (file1 == NULL)
    {
        printf("open error!\n");
        return;
    }
    fwrite(filename, strlen(filename), 1, file1);
    fwrite(",", strlen(","), 1, file1);
    fwrite(s, strlen(s), 1, file1);
    free(s);
    hipFree(k_val);
    hipFree(k_b);
    hipFree(k_x);
    hipFree(k_r);
    hipFree(k_d);
    hipFree(k_q);
    hipFree(k_alpha);
    hipFree(k_snew);
    hipFree(k_sold);
    hipFree(k_beta);
    hipFree(k_s0);
    hipFree(d_tile_ptr);
    hipFree(d_tile_columnidx);
    hipFree(d_csr_compressedIdx);
    hipFree(d_Blockcsr_Val);
    hipFree(d_Blockcsr_Ptr);
    hipFree(d_blkcoostylerowidx);
    hipFree(d_blkcoostylerowidx_colstart);
    hipFree(d_blkcoostylerowidx_colstop);
    hipFree(d_ptroffset1);
    hipFree(d_ptroffset2);
    hipFree(d_x);
    hipFree(d_y);
    free(matrix);
    free(ptroffset1);
    free(ptroffset2);
    free(y_golden);
    free(y);
    free(blkcoostylerowidx);
    free(blkcoostylerowidx_colstart);
    free(blkcoostylerowidx_colstop);
    free(tile_ptr);
    free(tile_columnidx);
    free(tile_nnz);
    free(csr_offset);
    free(csrptr_offset);
    free(Blockcsr_Val);
    free(Blockcsr_Val_Low);
    free(csr_compressedIdx);
    free(Blockcsr_Ptr);
}
int main(int argc, char **argv)
{
    char *filename = argv[1];
    int m, n, nnzR, isSymmetric;
    int *RowPtr;
    int *ColIdx;
    MAT_VAL_TYPE *Val;
    read_Dmatrix_32(&m, &n, &nnzR, &RowPtr, &ColIdx, &Val, &isSymmetric, filename);
    if(m!=n)
    {
        printf("unequal\n");
        return 0;
    }
    MAT_VAL_LOW_TYPE *Val_Low = (MAT_VAL_LOW_TYPE *)malloc(sizeof(MAT_VAL_LOW_TYPE) * nnzR);
    for (int i = 0; i < nnzR; i++)
    {
        Val_Low[i] = Val[i];
    }
    int ori = n;
    n = (n / BLOCK_SIZE) * BLOCK_SIZE;
    m = (m / BLOCK_SIZE) * BLOCK_SIZE;
    MAT_VAL_TYPE *X = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * (n));
    MAT_VAL_TYPE *Y = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * (m));
    MAT_VAL_TYPE *Y_golden = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * (m));
    memset(X, 0, sizeof(MAT_VAL_TYPE) * (n));
    memset(Y, 0, sizeof(MAT_VAL_TYPE) * (n));
    memset(Y_golden, 0, sizeof(MAT_VAL_TYPE) * (n));

    for (int i = 0; i < n; i++)
    {
        X[i] = 1;
    }
    int iter = 0;
    for (int i = 0; i < n; i++)
        for (int j = RowPtr[i]; j < RowPtr[i + 1]; j++)
            Y_golden[i] += Val[j] * X[ColIdx[j]];

    if(nnzR<10000)
    cg_solve_inc(RowPtr, ColIdx, Val, Val_Low, X, Y_golden, n, &iter, 10, 1e-5, filename, nnzR, ori);
    else if(nnzR<100000&&nnzR>=10000)
    cg_solve_sync(RowPtr, ColIdx, Val, Val_Low, X, Y_golden, n, &iter, 10, 1e-5, filename, nnzR, ori);
    else if(nnzR>=100000)
    cg_solve_reduce(RowPtr, ColIdx, Val, Val_Low, X, Y_golden, n, &iter, 10, 1e-5, filename, nnzR, ori);
}
